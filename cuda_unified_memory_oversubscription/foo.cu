
#include <hip/hip_runtime.h>
#include <cstdio>
#include <string>

#include <hip/hip_runtime_api.h>

void CheckCudaError(hipError_t err, int line) {
    if (err != hipSuccess) {
        printf("LINE: %d, %s\n", line, hipGetErrorString(err));
    }
}

#define CHECK(x) CheckCudaError((x), __LINE__)

int main (int argc, const char* argv[]) {
    if (argc < 3) {
        printf("usage:\n"
            "$ foo <itr, integer> <size, integer> <use 1)hipMemset or 0) memset, integer>\n"
            "$ foo 85 1000 1\n");
        return 1;
    }

    int itr = std::stoi(argv[1]);
    printf("%d times allocations\n", itr);

    size_t size = std::stoi(argv[2]);
    size *= 1024; // KB
    size *= 1024; // MB
    printf("allocate %zu B at a time\n", size);
    printf("allocate %zu B totally\n", size * itr);

    int use_cudamemset = std::stoi(argv[3]);
    printf("use %s to access the buffers\n",
        (use_cudamemset)?"hipMemset":"memset"
    );

    CHECK(hipSetDevice(0));
    for (int i = 0; i < itr; ++i) {
	void* ptr;
        size_t free, total;
        CHECK(hipMemGetInfo(&free, &total));
        printf("free: %zu, total: %zu\n", free, total);
        CHECK(hipMallocManaged((void**)&ptr, size));
        if (use_cudamemset) {
            CHECK(hipMemset(ptr, 0, size));
        } 
        else {
            memset(ptr, 0, size);
        }
    }
    while(1);
    return 0;
}
